#include "hip/hip_runtime.h"
#ifndef MEANSHIFT_CU
#define MEANSHIFT_CU

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include "vector_functions.hpp"
#include "hip/hip_vector_types.h"
#include "hip/hip_vector_types.h"
#include "hip/device_functions.h" 
#include "commonDefines.h"


#define MYASSERT(condition, ERROR) if (!(condition)) { printf("ERROR: %s \n", ERROR); return; }
#define rev_sqrt_two_pi 0.3989422804
#define rev_two_pi rev_sqrt_two_pi*rev_sqrt_two_pi

__device__ __host__ float gaussian_kernel(float dist2, float bandwidth) {
	const float rev_bandwidth = 1. / bandwidth;
	const float d2_frac_b2 = dist2 * rev_bandwidth * rev_bandwidth;
	float div = 1. / rev_two_pi * rev_bandwidth;
	float exp_ = div * expf(- 0.5 * d2_frac_b2);
	return exp_;
}

__global__ void cuda_MeanShift_SharedMemory_2D(float *X, const float *I, const float * originalPoints, const int N, const int dim) {

	__shared__ float tile[TILE_WIDTH][2];

	// for each pixel
	int tx = threadIdx.x;
	int row = blockIdx.x*blockDim.x + tx;

	float2 numerator = make_float2(0.0, 0.0);
	float denominator = 0.0;
	int it = row * dim;

	for (int tile_i = 0; tile_i < (N - 1) / TILE_WIDTH + 1; ++tile_i) {
		//loading phase - each thread load something into shared memory
		int row_t = tile_i * TILE_WIDTH + tx;

		int index = row_t * dim;
		if (row_t < N) {
			tile[tx][0] = originalPoints[index];
			tile[tx][1] = originalPoints[index + 1];
		}
		else {
			tile[tx][0] = 0.0;
			tile[tx][1] = 0.0;
		}
		__syncthreads();
		//end of loading into shared memory

		if (row < N) // only the threads inside the bounds do some computation
		{
			float2 x_i = make_float2(I[it], I[it + 1]); //load input point

			//computing phase
			for (int j = 0; j < TILE_WIDTH; ++j) {
				float2 x_j = make_float2(tile[j][0], tile[j][1]); //from shared memory
				float2 sub = x_i - x_j;
				float distance2 = dot(sub, sub);
				float weight = gaussian_kernel(distance2, BW);
				numerator += x_j * weight; //accumulating
				denominator += weight;

			}
		}
		__syncthreads();
		//end of computing phase for tile_ij
	}

	if (row < N) {
		//storing
		numerator /= denominator;
		X[it] = numerator.x;
		X[it + 1] = numerator.y;
	}

}

extern "C"
void cudaMeanShift_sharedMemory_2D_wrapper(float *X, const float *I, const float * originalPoints, const int N, const int vecDim, dim3 gridDim, dim3 blockDim) {
	cuda_MeanShift_SharedMemory_2D <<<gridDim, blockDim >>> (X, I, originalPoints, N, vecDim);
}

__global__ void cuda_MeanShift_2D(float *X, const float *I, const float * originalPoints, const int N, const int dim) {

	// for every pixel
	int tx = threadIdx.x;
	int row = blockIdx.x*blockDim.x + tx;

	float2 numerator = make_float2(0.0, 0.0);
	float denominator = 0.0;

	int it = row * dim;
	float2 y_i;
	if (row < N) {
		y_i = make_float2(I[it], I[it + 1]); //load input point

			//computing mean shift
			for (int j = 0; j < N; ++j) {
				float2 x_j = make_float2(originalPoints[j*dim], originalPoints[j*dim + 1]); //from central gpu memory
				float2 sub = y_i - x_j;
				float distance2 = dot(sub, sub);
				float weight = gaussian_kernel(distance2, BW);
				numerator += x_j * weight; //accumulating
				denominator += weight;
			}

		//storing
		numerator /= denominator;
		X[it] = numerator.x;
		X[it + 1] = numerator.y;
	}

}

extern "C"
void cudaMeanShift_2D_wrapper(float *X, const float *I, const float * originalPoints, const int N, const int vecDim, dim3 gridDim, dim3 blockDim) {
	cuda_MeanShift_2D <<<gridDim, blockDim >>> (X, I, originalPoints, N, vecDim);
}

#endif // !MEANSHIFT_CU
